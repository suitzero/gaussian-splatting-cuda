#include "hip/hip_runtime.h"
// kernels/newton_kernels.cu
#include "newton_kernels.cuh" // Now in the same directory
#include "kernels/ssim.cuh"   // For fusedssim, fusedssim_backward C++ functions
#include <hip/hip_runtime.h> // Includes hip/hip_vector_types.h for ::float3, ::float2
#include <>
#include <torch/torch.h> // For AT_ASSERTM
#include <cmath> // For fabsf, sqrtf, etc.

// Basic CUDA utilities (normally in a separate header)
#define CUDA_CHECK(status) AT_ASSERTM(status == hipSuccess, hipGetErrorString(status))

constexpr int CUDA_NUM_THREADS = 256; // Default number of threads per block
inline int GET_BLOCKS(const int N) {
    return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}


// --- CUDA Math Helper Functions ---
// Using standard CUDA vector types (e.g., ::float3, ::float2 from hip/hip_vector_types.h)
// and basic operations.
namespace CudaMath {

// Vector operations using ::float3, ::float2
__device__ __forceinline__ ::float3 add_float3(const ::float3& a, const ::float3& b) {
    return ::make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ __forceinline__ ::float3 sub_float3(const ::float3& a, const ::float3& b) {
    return ::make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ __forceinline__ ::float3 mul_float3_scalar(const ::float3& v, float s) {
    return ::make_float3(v.x * s, v.y * s, v.z * s);
}

__device__ __forceinline__ ::float3 div_float3_scalar(const ::float3& v, float s) {
    float inv_s = 1.0f / (s + 1e-8f); // Add epsilon for stability
    return ::make_float3(v.x * inv_s, v.y * inv_s, v.z * inv_s);
}

__device__ __forceinline__ float dot_product(const ::float3& a, const ::float3& b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ __forceinline__ ::float3 cross_product(const ::float3& a, const ::float3& b) {
    return ::make_float3(a.y * b.z - a.z * b.y,
                         a.z * b.x - a.x * b.z,
                         a.x * b.y - a.y * b.x);
}

__device__ __forceinline__ float length_sq_float3(const ::float3& v) {
    return v.x * v.x + v.y * v.y + v.z * v.z;
}

__device__ __forceinline__ float length_float3(const ::float3& v) {
    return sqrtf(length_sq_float3(v));
}

__device__ __forceinline__ ::float3 normalize_vec3(const ::float3& v) {
    float l = length_float3(v);
    return div_float3_scalar(v, l);
}

// Matrix operations (assuming row-major for M)
__device__ __forceinline__ ::float3 mul_mat3_vec3(const float* M, const ::float3& v) {
    ::float3 r;
    r.x = M[0] * v.x + M[1] * v.y + M[2] * v.z;
    r.y = M[3] * v.x + M[4] * v.y + M[5] * v.z;
    r.z = M[6] * v.x + M[7] * v.y + M[8] * v.z;
    return r;
}

__device__ __forceinline__ void mat3_transpose_inplace(float* M) {
    float temp;
    temp = M[1]; M[1] = M[3]; M[3] = temp;
    temp = M[2]; M[2] = M[6]; M[6] = temp;
    temp = M[5]; M[5] = M[7]; M[7] = temp;
}

__device__ __forceinline__ void outer_product_3x3(const ::float3& a, const ::float3& b, float* out_M) {
    out_M[0] = a.x * b.x; out_M[1] = a.x * b.y; out_M[2] = a.x * b.z;
    out_M[3] = a.y * b.x; out_M[4] = a.y * b.y; out_M[5] = a.y * b.z;
    out_M[6] = a.z * b.x; out_M[7] = a.z * b.y; out_M[8] = a.z * b.z;
}

__device__ __forceinline__ void mul_mat4_vec4(const float* PW, const float* p_k_h, float* result) {
    for (int i = 0; i < 4; ++i) {
        result[i] = 0;
        for (int j = 0; j < 4; ++j) {
            result[i] += PW[i * 4 + j] * p_k_h[j];
        }
    }
}

__device__ __forceinline__ void mat_mul_vec(const float* M, const float* v, float* out) {
    for (int i = 0; i < 3; ++i) {
        out[i] = 0;
        for (int j = 0; j < 3; ++j) {
            out[i] += M[i * 3 + j] * v[j];
        }
    }
}

__device__ __forceinline__ void mat_mul_mat(const float* A, const float* B, float* C,
                                         int A_rows, int A_cols_B_rows, int B_cols) {
    for (int i = 0; i < A_rows; ++i) {
        for (int j = 0; j < B_cols; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < A_cols_B_rows; ++k) {
                sum += A[i * A_cols_B_rows + k] * B[k * B_cols + j];
            }
            C[i * B_cols + j] = sum;
        }
    }
}

} // namespace CudaMath


// --- Projection Derivative Helper Functions ---
namespace ProjectionDerivs {

__device__ __forceinline__ void compute_h_vec(const ::float3& p_k, const float* PW, float* h_vec4) {
    float p_k_h[4] = {p_k.x, p_k.y, p_k.z, 1.0f};
    CudaMath::mul_mat4_vec4(PW, p_k_h, h_vec4);
}

__device__ __forceinline__ void compute_projection_jacobian(
    const float* PW, float W_I_t, float H_I_t,
    const float* h_vec4, float* jacobian_out_2x3
) {
    float hx = h_vec4[0];
    float hy = h_vec4[1];
    float hw = h_vec4[3];
    float inv_hw = 1.0f / (hw + 1e-8f);

    float term_x_coeff = W_I_t / 2.0f;
    float term_y_coeff = H_I_t / 2.0f;

    for (int j = 0; j < 3; ++j) {
        jacobian_out_2x3[0 * 3 + j] = term_x_coeff * (inv_hw * PW[0 * 4 + j] - (hx * inv_hw * inv_hw) * PW[3 * 4 + j]);
    }
    for (int j = 0; j < 3; ++j) {
        jacobian_out_2x3[1 * 3 + j] = term_y_coeff * (inv_hw * PW[1 * 4 + j] - (hy * inv_hw * inv_hw) * PW[3 * 4 + j]);
    }
}

__device__ __forceinline__ void compute_projection_hessian(
    const float* PW, float W_I_t, float H_I_t,
    const float* h_vec4,
    float* hessian_out_pi_x_3x3, float* hessian_out_pi_y_3x3
) {
    float hx = h_vec4[0];
    float hy = h_vec4[1];
    float hw = h_vec4[3];
    float inv_hw_sq = 1.0f / (hw * hw + 1e-8f);

    ::float3 PW0_vec = ::make_float3(PW[0], PW[1], PW[2]);
    ::float3 PW1_vec = ::make_float3(PW[4], PW[5], PW[6]);
    ::float3 PW3_vec = ::make_float3(PW[12], PW[13], PW[14]);

    float PW3_outer_PW3[9];
    CudaMath::outer_product_3x3(PW3_vec, PW3_vec, PW3_outer_PW3);
    float PW3_outer_PW0[9];
    CudaMath::outer_product_3x3(PW3_vec, PW0_vec, PW3_outer_PW0);
    float PW3_outer_PW1[9];
    CudaMath::outer_product_3x3(PW3_vec, PW1_vec, PW3_outer_PW1);

    float factor_x1 = W_I_t * (2.0f * hx / (hw*hw*hw + 1e-9f));
    float factor_x2 = W_I_t * (-1.0f * inv_hw_sq);
    for (int i = 0; i < 9; ++i) {
        float term1_x = factor_x1 * PW3_outer_PW3[i];
        int row = i / 3;
        int col = i % 3;
        float term2_x = factor_x2 * (PW3_outer_PW0[i] + PW3_outer_PW0[col * 3 + row]);
        hessian_out_pi_x_3x3[i] = term1_x + term2_x;
    }

    float factor_y1 = H_I_t * (2.0f * hy / (hw*hw*hw + 1e-9f));
    float factor_y2 = H_I_t * (-1.0f * inv_hw_sq);
    for (int i = 0; i < 9; ++i) {
        float term1_y = factor_y1 * PW3_outer_PW3[i];
        int row = i / 3;
        int col = i % 3;
        float term2_y = factor_y2 * (PW3_outer_PW1[i] + PW3_outer_PW1[col * 3 + row]);
        hessian_out_pi_y_3x3[i] = term1_y + term2_y;
    }
}
} // namespace ProjectionDerivs

// --- SH Basis and Color Derivative Helper Functions ---
namespace SHDerivs {

__device__ __forceinline__ void eval_sh_basis_up_to_degree3(
    int degree, const ::float3& r_k_normalized, float* basis_out
) {
    float x = r_k_normalized.x;
    float y = r_k_normalized.y;
    float z = r_k_normalized.z;
    basis_out[0] = 0.2820947917738781f;
    if (degree == 0) return;
    basis_out[1] = -0.48860251190292f * y;
    basis_out[2] =  0.48860251190292f * z;
    basis_out[3] = -0.48860251190292f * x;
    if (degree == 1) return;
    float x2 = x*x; float y2 = y*y; float z2 = z*z;
    basis_out[4] =  0.5462742152960395f * (2.f * x * y);
    basis_out[5] = -1.092548430592079f * y * z;
    basis_out[6] =  0.3153915652525201f * (3.f * z2 - 1.f);
    basis_out[7] = -1.092548430592079f * x * z;
    basis_out[8] =  0.5462742152960395f * (x2 - y2);
    if (degree == 2) return;
    float fC1 = x2 - y2; float fS1 = 2.f * x * y;
    float fC2 = x * fC1 - y * fS1; float fS2 = y * fC1 + x * fS1;
    float fTmp0C = -2.285228997322329f * z2 + 0.4570457994644658f;
    float fTmp1B = 1.445305721320277f * z;
    basis_out[9]  = -0.5900435899266435f * fS2;
    basis_out[10] = fTmp1B * fS1;
    basis_out[11] = fTmp0C * y;
    basis_out[12] = z * (1.865881662950577f * z2 - 1.119528997770346f);
    basis_out[13] = fTmp0C * x;
    basis_out[14] = fTmp1B * fC1;
    basis_out[15] = -0.5900435899266435f * fC2;
}

__device__ __forceinline__ void compute_drk_dpk(
    const ::float3& r_k_normalized, float r_k_norm, float* drk_dpk_out_3x3
) {
    float inv_r_k_norm = 1.0f / (r_k_norm + 1e-8f);
    drk_dpk_out_3x3[0] = 1.0f; drk_dpk_out_3x3[1] = 0.0f; drk_dpk_out_3x3[2] = 0.0f;
    drk_dpk_out_3x3[3] = 0.0f; drk_dpk_out_3x3[4] = 1.0f; drk_dpk_out_3x3[5] = 0.0f;
    drk_dpk_out_3x3[6] = 0.0f; drk_dpk_out_3x3[7] = 0.0f; drk_dpk_out_3x3[8] = 1.0f;
    drk_dpk_out_3x3[0] -= r_k_normalized.x * r_k_normalized.x;
    drk_dpk_out_3x3[1] -= r_k_normalized.x * r_k_normalized.y;
    drk_dpk_out_3x3[2] -= r_k_normalized.x * r_k_normalized.z;
    drk_dpk_out_3x3[3] -= r_k_normalized.y * r_k_normalized.x;
    drk_dpk_out_3x3[4] -= r_k_normalized.y * r_k_normalized.y;
    drk_dpk_out_3x3[5] -= r_k_normalized.y * r_k_normalized.z;
    drk_dpk_out_3x3[6] -= r_k_normalized.z * r_k_normalized.x;
    drk_dpk_out_3x3[7] -= r_k_normalized.z * r_k_normalized.y;
    drk_dpk_out_3x3[8] -= r_k_normalized.z * r_k_normalized.z;
    for (int i = 0; i < 9; ++i) {
        drk_dpk_out_3x3[i] *= inv_r_k_norm;
    }
}

__device__ __forceinline__ void compute_dphi_drk_up_to_degree3(
    int degree, const ::float3& r_k_normalized, float* dPhi_drk_out
) {
    float x = r_k_normalized.x; float y = r_k_normalized.y; float z = r_k_normalized.z;
    float x2 = x*x; float y2 = y*y; float z2 = z*z;
    dPhi_drk_out[0*3 + 0] = 0.0f; dPhi_drk_out[0*3 + 1] = 0.0f; dPhi_drk_out[0*3 + 2] = 0.0f;
    if (degree == 0) return;
    dPhi_drk_out[1*3 + 0] = 0.0f; dPhi_drk_out[1*3 + 1] = -0.48860251190292f; dPhi_drk_out[1*3 + 2] = 0.0f;
    dPhi_drk_out[2*3 + 0] = 0.0f; dPhi_drk_out[2*3 + 1] = 0.0f; dPhi_drk_out[2*3 + 2] = 0.48860251190292f;
    dPhi_drk_out[3*3 + 0] = -0.48860251190292f; dPhi_drk_out[3*3 + 1] = 0.0f; dPhi_drk_out[3*3 + 2] = 0.0f;
    if (degree == 1) return;
    const float C2_0_val = 1.092548430592079f;
    const float C2_1_val = -1.092548430592079f;
    const float C2_2_val_scaled = 0.9461746957575601f;
    dPhi_drk_out[4*3 + 0] = (C2_0_val/2.f) * y; dPhi_drk_out[4*3 + 1] = (C2_0_val/2.f) * x; dPhi_drk_out[4*3 + 2] = 0.0f;
    dPhi_drk_out[5*3 + 0] = 0.0f; dPhi_drk_out[5*3 + 1] = C2_1_val * z; dPhi_drk_out[5*3 + 2] = C2_1_val * y;
    dPhi_drk_out[6*3 + 0] = 0.0f; dPhi_drk_out[6*3 + 1] = 0.0f; dPhi_drk_out[6*3 + 2] = (C2_2_val_scaled/3.f) * (6.f * z);
    dPhi_drk_out[7*3 + 0] = C2_1_val * z; dPhi_drk_out[7*3 + 1] = 0.0f; dPhi_drk_out[7*3 + 2] = C2_1_val * x;
    dPhi_drk_out[8*3 + 0] = (C2_0_val/2.f) * (2.f * x); dPhi_drk_out[8*3 + 1] = (C2_0_val/2.f) * (-2.f * y); dPhi_drk_out[8*3 + 2] = 0.0f;
    if (degree == 2) return;
    const float K9_val = -0.5900435899266435f;
    const float K10_z_coeff_val = 1.445305721320277f;
    const float K11_a_coeff_val = -2.285228997322329f;
    const float K11_b_coeff_val = 0.4570457994644658f;
    const float K12_a_coeff_val = 1.865881662950577f;
    const float K12_b_coeff_val = -1.119528997770346f;
    const float K15_val = -0.5900435899266435f;
    dPhi_drk_out[9*3 + 0] = K9_val * (6.f*x*y);
    dPhi_drk_out[9*3 + 1] = K9_val * (3.f*x2 - 3.f*y2);
    dPhi_drk_out[9*3 + 2] = 0.0f;
    dPhi_drk_out[10*3 + 0] = K10_z_coeff_val * z * (2.f*y);
    dPhi_drk_out[10*3 + 1] = K10_z_coeff_val * z * (2.f*x);
    dPhi_drk_out[10*3 + 2] = K10_z_coeff_val * (2.f*x*y);
    dPhi_drk_out[11*3 + 0] = 0.0f;
    dPhi_drk_out[11*3 + 1] = K11_a_coeff_val * z2 + K11_b_coeff_val;
    dPhi_drk_out[11*3 + 2] = K11_a_coeff_val * y * (2.f*z);
    dPhi_drk_out[12*3 + 0] = 0.0f;
    dPhi_drk_out[12*3 + 1] = 0.0f;
    dPhi_drk_out[12*3 + 2] = K12_a_coeff_val * 3.f*z2 + K12_b_coeff_val;
    dPhi_drk_out[13*3 + 0] = K11_a_coeff_val * z2 + K11_b_coeff_val;
    dPhi_drk_out[13*3 + 1] = 0.0f;
    dPhi_drk_out[13*3 + 2] = K11_a_coeff_val * x * (2.f*z);
    dPhi_drk_out[14*3 + 0] = K10_z_coeff_val * z * (2.f*x);
    dPhi_drk_out[14*3 + 1] = K10_z_coeff_val * z * (-2.f*y);
    dPhi_drk_out[14*3 + 2] = K10_z_coeff_val * (x2 - y2);
    dPhi_drk_out[15*3 + 0] = K15_val * (3.f*x2 - 3.f*y2);
    dPhi_drk_out[15*3 + 1] = K15_val * (-6.f*x*y);
    dPhi_drk_out[15*3 + 2] = 0.0f;
}

__device__ __forceinline__ void compute_sh_color_jacobian_single_channel(
    const float* sh_coeffs_single_channel, const float* sh_basis_values,
    const float* dPhi_drk, const float* drk_dpk,
    int num_basis_coeffs, float* jac_out_3
) {
    float M_prod[16*3];
    CudaMath::mat_mul_mat(dPhi_drk, drk_dpk, M_prod, num_basis_coeffs, 3, 3);
    jac_out_3[0] = 0.0f; jac_out_3[1] = 0.0f; jac_out_3[2] = 0.0f;
    for (int i = 0; i < num_basis_coeffs; ++i) {
        float v_i = sh_basis_values[i] * sh_coeffs_single_channel[i];
        jac_out_3[0] += v_i * M_prod[i * 3 + 0];
        jac_out_3[1] += v_i * M_prod[i * 3 + 1];
        jac_out_3[2] += v_i * M_prod[i * 3 + 2];
    }
}
} // namespace SHDerivs

// --- KERNEL DEFINITIONS ---

__device__ __forceinline__ void get_projected_cov2d_and_derivs_placeholder(
    const ::float3& p_k_world,
    const float* scales_k, const float* rotations_k,
    const float* view_matrix, const float* proj_matrix,
    const float* jacobian_d_pi_d_pk,
    float img_W, float img_H,
    float* cov2d_sym, float* inv_cov2d_sym, float* det_cov2d,
    float* d_Gk_d_pik, float* d2_Gk_d_pik2
) {
    cov2d_sym[0] = 1.0f; cov2d_sym[1] = 0.0f; cov2d_sym[2] = 1.0f;
    inv_cov2d_sym[0] = 1.0f; inv_cov2d_sym[1] = 0.0f; inv_cov2d_sym[2] = 1.0f;
    *det_cov2d = 1.0f;
    if (d_Gk_d_pik) {
        d_Gk_d_pik[0] = 0.0f; d_Gk_d_pik[1] = 0.0f;
    }
    if (d2_Gk_d_pik2) {
        d2_Gk_d_pik2[0] = -1.0f * inv_cov2d_sym[0];
        d2_Gk_d_pik2[1] = -1.0f * inv_cov2d_sym[1];
        d2_Gk_d_pik2[2] = -1.0f * inv_cov2d_sym[2];
    }
}

__global__ void compute_l1l2_loss_derivatives_kernel(
    const float* rendered_image, const float* gt_image, bool use_l2_loss_term,
    float inv_N_pixels, float* out_dL_dc_l1l2, float* out_d2L_dc2_diag_l1l2,
    int H, int W, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = H * W * C;
    if (idx >= total_elements) return;
    float diff = rendered_image[idx] - gt_image[idx];
    if (use_l2_loss_term) {
        out_dL_dc_l1l2[idx] = inv_N_pixels * 2.f * diff;
        out_d2L_dc2_diag_l1l2[idx] = inv_N_pixels * 2.f;
    } else {
        out_dL_dc_l1l2[idx] = inv_N_pixels * ((diff > 1e-6f) ? 1.f : ((diff < -1e-6f) ? -1.f : 0.f));
        out_d2L_dc2_diag_l1l2[idx] = 0.f;
    }
}

__global__ void compute_position_hessian_components_kernel(
    int H_img, int W_img, int C_img,
    int P_total,
    const float* means_3d_all,
    const float* scales_all,
    const float* rotations_all,
    const float* opacities_all,
    const float* shs_all,
    int sh_degree,
    int sh_coeffs_per_color_channel,
    const float* view_matrix,
    const float* projection_matrix_for_jacobian,
    const float* cam_pos_world,
    const bool* visibility_mask_for_model,
    const float* dL_dc_pixelwise,
    const float* d2L_dc2_diag_pixelwise,
    int num_output_gaussians,
    float* H_p_output_packed,
    float* grad_p_output,
    const int* output_index_map,
    bool debug_prints_enabled
) {
    int p_idx_total = blockIdx.x * blockDim.x + threadIdx.x;

    if (p_idx_total >= P_total) return;
    if (!visibility_mask_for_model[p_idx_total]) return;

    int output_idx = output_index_map[p_idx_total];
    if (output_idx < 0 || output_idx >= num_output_gaussians) return;

    ::float3 pk_vec3 = ::make_float3(
        means_3d_all[p_idx_total * 3 + 0],
        means_3d_all[p_idx_total * 3 + 1],
        means_3d_all[p_idx_total * 3 + 2]);

    const float* scales_k = scales_all + p_idx_total * 3;
    const float* rotations_k = rotations_all + p_idx_total * 4;
    float opacity_k = opacities_all[p_idx_total];
    const float* sh_coeffs_k_all_channels = shs_all + p_idx_total * sh_coeffs_per_color_channel * 3;

    ::float3 cam_pos_world_vec3 = ::make_float3(cam_pos_world[0], cam_pos_world[1], cam_pos_world[2]);

    ::float3 view_dir_to_pk_unnormalized = CudaMath::sub_float3(pk_vec3, cam_pos_world_vec3);
    float r_k_norm = CudaMath::length_float3(view_dir_to_pk_unnormalized);
    ::float3 r_k_normalized = CudaMath::div_float3_scalar(view_dir_to_pk_unnormalized, r_k_norm);

    float proj_view_matrix[16];
    CudaMath::mat_mul_mat(projection_matrix_for_jacobian, view_matrix, proj_view_matrix, 4, 4, 4);

    float h_vec4_data[4];
    ProjectionDerivs::compute_h_vec(pk_vec3, proj_view_matrix, h_vec4_data);

    float d_pi_d_pk_data[2*3];
    ProjectionDerivs::compute_projection_jacobian(proj_view_matrix, (float)W_img, (float)H_img, h_vec4_data, d_pi_d_pk_data);

    float d2_pi_d_pk2_x_data[3*3];
    float d2_pi_d_pk2_y_data[3*3];
    ProjectionDerivs::compute_projection_hessian(proj_view_matrix, (float)W_img, (float)H_img, h_vec4_data, d2_pi_d_pk2_x_data, d2_pi_d_pk2_y_data);

    float sh_basis_eval_data[16];
    SHDerivs::eval_sh_basis_up_to_degree3(sh_degree, r_k_normalized, sh_basis_eval_data);

    float d_rk_d_pk_data[3*3];
    SHDerivs::compute_drk_dpk(r_k_normalized, r_k_norm, d_rk_d_pk_data);

    float d_phi_d_rk_data[16*3];
    SHDerivs::compute_dphi_drk_up_to_degree3(sh_degree, r_k_normalized, d_phi_d_rk_data);

    ::float3 d_c_bar_R_d_pk_val, d_c_bar_G_d_pk_val, d_c_bar_B_d_pk_val;
    float sh_coeffs_k_R[16], sh_coeffs_k_G[16], sh_coeffs_k_B[16];
    for(int i=0; i<sh_coeffs_per_color_channel; ++i) {
        sh_coeffs_k_R[i] = sh_coeffs_k_all_channels[i*3 + 0];
        sh_coeffs_k_G[i] = sh_coeffs_k_all_channels[i*3 + 1];
        sh_coeffs_k_B[i] = sh_coeffs_k_all_channels[i*3 + 2];
    }

    SHDerivs::compute_sh_color_jacobian_single_channel(sh_coeffs_k_R, sh_basis_eval_data, d_phi_d_rk_data, d_rk_d_pk_data, sh_coeffs_per_color_channel, &d_c_bar_R_d_pk_val.x);
    SHDerivs::compute_sh_color_jacobian_single_channel(sh_coeffs_k_G, sh_basis_eval_data, d_phi_d_rk_data, d_rk_d_pk_data, sh_coeffs_per_color_channel, &d_c_bar_G_d_pk_val.x);
    SHDerivs::compute_sh_color_jacobian_single_channel(sh_coeffs_k_B, sh_basis_eval_data, d_phi_d_rk_data, d_rk_d_pk_data, sh_coeffs_per_color_channel, &d_c_bar_B_d_pk_val.x);

    ::float3 g_p_k_accum_val = ::make_float3(0.f, 0.f, 0.f);
    float H_p_k_accum_symm[6] = {0.f, 0.f, 0.f, 0.f, 0.f, 0.f};

    for (int r = 0; r < H_img; ++r) {
        for (int c = 0; c < W_img; ++c) {
            float pixel_ndc_x = (2.0f * (c + 0.5f) / W_img - 1.0f);
            float pixel_ndc_y = (2.0f * (r + 0.5f) / H_img - 1.0f);

            float pi_k_ndc_x_unscaled = h_vec4_data[0] / (h_vec4_data[3] + 1e-7f);
            float pi_k_ndc_y_unscaled = h_vec4_data[1] / (h_vec4_data[3] + 1e-7f);

            ::float2 diff_ndc_val = ::make_float2(pi_k_ndc_x_unscaled - pixel_ndc_x, pi_k_ndc_y_unscaled - pixel_ndc_y);

            float cov2d_sym_data[3], inv_cov2d_sym_data[3], det_cov2d_data;
            float d_Gk_d_pik_data[2];
            float d2_Gk_d_pik2_data[3];

            get_projected_cov2d_and_derivs_placeholder(pk_vec3, scales_k, rotations_k,
                                                       view_matrix, projection_matrix_for_jacobian,
                                                       d_pi_d_pk_data, (float)W_img, (float)H_img,
                                                       cov2d_sym_data, inv_cov2d_sym_data, &det_cov2d_data,
                                                       nullptr, nullptr);

            float G_k_pixel = expf(-0.5f * (diff_ndc_val.x*diff_ndc_val.x*inv_cov2d_sym_data[0] +
                                            2*diff_ndc_val.x*diff_ndc_val.y*inv_cov2d_sym_data[1] +
                                            diff_ndc_val.y*diff_ndc_val.y*inv_cov2d_sym_data[2]));
            if (det_cov2d_data <= 1e-7f) G_k_pixel = 0.f;

            if (G_k_pixel < 1e-4f) continue;

            ::float2 sigma_inv_diff_val;
            sigma_inv_diff_val.x = inv_cov2d_sym_data[0]*diff_ndc_val.x + inv_cov2d_sym_data[1]*diff_ndc_val.y;
            sigma_inv_diff_val.y = inv_cov2d_sym_data[1]*diff_ndc_val.x + inv_cov2d_sym_data[2]*diff_ndc_val.y;
            d_Gk_d_pik_data[0] = -G_k_pixel * sigma_inv_diff_val.x;
            d_Gk_d_pik_data[1] = -G_k_pixel * sigma_inv_diff_val.y;

            d2_Gk_d_pik2_data[0] = G_k_pixel * (sigma_inv_diff_val.x * sigma_inv_diff_val.x - inv_cov2d_sym_data[0]);
            d2_Gk_d_pik2_data[1] = G_k_pixel * (sigma_inv_diff_val.x * sigma_inv_diff_val.y - inv_cov2d_sym_data[1]);
            d2_Gk_d_pik2_data[2] = G_k_pixel * (sigma_inv_diff_val.y * sigma_inv_diff_val.y - inv_cov2d_sym_data[2]);


            float alpha_k_pixel = opacity_k * G_k_pixel;

            ::float3 c_bar_k_rgb_val;
            c_bar_k_rgb_val.x =0; for(int i=0; i<sh_coeffs_per_color_channel; ++i) c_bar_k_rgb_val.x += sh_coeffs_k_R[i] * sh_basis_eval_data[i];
            c_bar_k_rgb_val.y =0; for(int i=0; i<sh_coeffs_per_color_channel; ++i) c_bar_k_rgb_val.y += sh_coeffs_k_G[i] * sh_basis_eval_data[i];
            c_bar_k_rgb_val.z =0; for(int i=0; i<sh_coeffs_per_color_channel; ++i) c_bar_k_rgb_val.z += sh_coeffs_k_B[i] * sh_basis_eval_data[i];


            ::float3 d_c_final_d_Gk_val = CudaMath::mul_float3_scalar(c_bar_k_rgb_val, opacity_k);

            ::float3 d_Gk_d_pk_chain_val;
            d_Gk_d_pk_chain_val.x = d_Gk_d_pik_data[0] * d_pi_d_pk_data[0*3+0] + d_Gk_d_pik_data[1] * d_pi_d_pk_data[1*3+0];
            d_Gk_d_pk_chain_val.y = d_Gk_d_pik_data[0] * d_pi_d_pk_data[0*3+1] + d_Gk_d_pik_data[1] * d_pi_d_pk_data[1*3+1];
            d_Gk_d_pk_chain_val.z = d_Gk_d_pik_data[0] * d_pi_d_pk_data[0*3+2] + d_Gk_d_pik_data[1] * d_pi_d_pk_data[1*3+2];

            ::float3 J_c_pk_R_val, J_c_pk_G_val, J_c_pk_B_val;
            J_c_pk_R_val = CudaMath::add_float3(CudaMath::mul_float3_scalar(d_c_bar_R_d_pk_val, alpha_k_pixel), CudaMath::mul_float3_scalar(d_Gk_d_pk_chain_val, d_c_final_d_Gk_val.x));
            J_c_pk_G_val = CudaMath::add_float3(CudaMath::mul_float3_scalar(d_c_bar_G_d_pk_val, alpha_k_pixel), CudaMath::mul_float3_scalar(d_Gk_d_pk_chain_val, d_c_final_d_Gk_val.y));
            J_c_pk_B_val = CudaMath::add_float3(CudaMath::mul_float3_scalar(d_c_bar_B_d_pk_val, alpha_k_pixel), CudaMath::mul_float3_scalar(d_Gk_d_pk_chain_val, d_c_final_d_Gk_val.z));

            int pixel_idx_flat = (r * W_img + c) * C_img;
            ::float3 dL_dc_val_pixel = ::make_float3(dL_dc_pixelwise[pixel_idx_flat+0], dL_dc_pixelwise[pixel_idx_flat+1], dL_dc_pixelwise[pixel_idx_flat+2]);
            ::float3 d2L_dc2_diag_val_pixel = ::make_float3(d2L_dc2_diag_pixelwise[pixel_idx_flat+0], d2L_dc2_diag_pixelwise[pixel_idx_flat+1], d2L_dc2_diag_pixelwise[pixel_idx_flat+2]);

            g_p_k_accum_val.x += J_c_pk_R_val.x * dL_dc_val_pixel.x + J_c_pk_G_val.x * dL_dc_val_pixel.y + J_c_pk_B_val.x * dL_dc_val_pixel.z;
            g_p_k_accum_val.y += J_c_pk_R_val.y * dL_dc_val_pixel.x + J_c_pk_G_val.y * dL_dc_val_pixel.y + J_c_pk_B_val.y * dL_dc_val_pixel.z;
            g_p_k_accum_val.z += J_c_pk_R_val.z * dL_dc_val_pixel.x + J_c_pk_G_val.z * dL_dc_val_pixel.y + J_c_pk_B_val.z * dL_dc_val_pixel.z;

            H_p_k_accum_symm[0] += J_c_pk_R_val.x * d2L_dc2_diag_val_pixel.x * J_c_pk_R_val.x + J_c_pk_G_val.x * d2L_dc2_diag_val_pixel.y * J_c_pk_G_val.x + J_c_pk_B_val.x * d2L_dc2_diag_val_pixel.z * J_c_pk_B_val.x;
            H_p_k_accum_symm[1] += J_c_pk_R_val.x * d2L_dc2_diag_val_pixel.x * J_c_pk_R_val.y + J_c_pk_G_val.x * d2L_dc2_diag_val_pixel.y * J_c_pk_G_val.y + J_c_pk_B_val.x * d2L_dc2_diag_val_pixel.z * J_c_pk_B_val.y;
            H_p_k_accum_symm[2] += J_c_pk_R_val.x * d2L_dc2_diag_val_pixel.x * J_c_pk_R_val.z + J_c_pk_G_val.x * d2L_dc2_diag_val_pixel.y * J_c_pk_G_val.z + J_c_pk_B_val.x * d2L_dc2_diag_val_pixel.z * J_c_pk_B_val.z;
            H_p_k_accum_symm[3] += J_c_pk_R_val.y * d2L_dc2_diag_val_pixel.x * J_c_pk_R_val.y + J_c_pk_G_val.y * d2L_dc2_diag_val_pixel.y * J_c_pk_G_val.y + J_c_pk_B_val.y * d2L_dc2_diag_val_pixel.z * J_c_pk_B_val.y;
            H_p_k_accum_symm[4] += J_c_pk_R_val.y * d2L_dc2_diag_val_pixel.x * J_c_pk_R_val.z + J_c_pk_G_val.y * d2L_dc2_diag_val_pixel.y * J_c_pk_G_val.z + J_c_pk_B_val.y * d2L_dc2_diag_val_pixel.z * J_c_pk_B_val.z;
            H_p_k_accum_symm[5] += J_c_pk_R_val.z * d2L_dc2_diag_val_pixel.x * J_c_pk_R_val.z + J_c_pk_G_val.z * d2L_dc2_diag_val_pixel.y * J_c_pk_G_val.z + J_c_pk_B_val.z * d2L_dc2_diag_val_pixel.z * J_c_pk_B_val.z;
        }
    }

    grad_p_output[output_idx * 3 + 0] = g_p_k_accum_val.x;
    grad_p_output[output_idx * 3 + 1] = g_p_k_accum_val.y;
    grad_p_output[output_idx * 3 + 2] = g_p_k_accum_val.z;

    for(int i=0; i<6; ++i) {
        H_p_output_packed[output_idx * 6 + i] = H_p_k_accum_symm[i];
    }
}

// Kernel for projecting Hessian and Gradient
__global__ void project_position_hessian_gradient_kernel(
    int num_visible_gaussians,
    const float* H_p_packed_input, // [N_vis, 6] (Hpxx, Hpxy, Hpxz, Hpyy, Hpyz, Hpzz)
    const float* grad_p_input,     // [N_vis, 3]
    const float* means_3d_visible, // [N_vis, 3]
    const float* view_matrix,      // [16] (col-major or row-major assumed by caller)
    const float* cam_pos_world,    // [3]
    float* out_H_v_packed,         // [N_vis, 3] (Hvxx, Hvxy, Hvyy)
    float* out_grad_v) {           // [N_vis, 2]

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_visible_gaussians) return;

    // 1. Calculate r_k = p_k - cam_pos_world (view vector from camera to point)
    //    It's often more convenient to use the camera's Z-axis (view direction) and up/right vectors.
    //    The paper's U_k = [u_x, u_y] forms a 2D basis perpendicular to r_k.
    //    Let r_k = means_3d_visible[idx*3+c] - cam_pos_world[c]
    //    This r_k is pointing from camera to Gaussian.
    //    The paper mentions "camera's look at r_k". This usually means r_k is the direction vector.
    //    Let's assume view_matrix gives camera orientation. view_matrix[2], view_matrix[6], view_matrix[10] is cam Z axis (if row-major).
    //    Let world_z_vec = {view_matrix[2], view_matrix[6], view_matrix[10]} (camera's forward vector)
    //    Let world_y_vec = {view_matrix[1], view_matrix[5], view_matrix[9]} (camera's up vector)
    //    Let world_x_vec = {view_matrix[0], view_matrix[4], view_matrix[8]} (camera's right vector)

    // Simplified U_k: use camera's X and Y axes in world space as u_x, u_y.
    // This assumes planar adjustment aligned with camera's own axes.
    // Paper Eq 14 is more complex: u_y = (r_k outer_prod r_k)[0,1,0]^T / norm(...)
    // This implies r_k is used to define the plane.
    // For now, let u_x = camera right, u_y = camera up. This is a common simplification for screen-space operations.
    float ux[3] = {view_matrix[0], view_matrix[4], view_matrix[8]}; // Camera Right
    float uy[3] = {view_matrix[1], view_matrix[5], view_matrix[9]}; // Camera Up

    // Project gradient: g_v = U^T g_p
    // g_v[0] = ux . grad_p_input[idx*3+c]
    // g_v[1] = uy . grad_p_input[idx*3+c]
    out_grad_v[idx*2 + 0] = ux[0]*grad_p_input[idx*3+0] + ux[1]*grad_p_input[idx*3+1] + ux[2]*grad_p_input[idx*3+2];
    out_grad_v[idx*2 + 1] = uy[0]*grad_p_input[idx*3+0] + uy[1]*grad_p_input[idx*3+1] + uy[2]*grad_p_input[idx*3+2];

    // Project Hessian: H_v = U^T H_p U
    // H_p matrix from packed:
    // [ H00 H01 H02 ]
    // [ H01 H11 H12 ]
    // [ H02 H12 H22 ]
    // H_p_packed_input = [H00, H01, H02, H11, H12, H22]
    const float* Hp = &H_p_packed_input[idx*6];
    float Hpu_x[3]; // H_p * u_x
    Hpu_x[0] = Hp[0]*ux[0] + Hp[1]*ux[1] + Hp[2]*ux[2];
    Hpu_x[1] = Hp[1]*ux[0] + Hp[3]*ux[1] + Hp[4]*ux[2];
    Hpu_x[2] = Hp[2]*ux[0] + Hp[4]*ux[1] + Hp[5]*ux[2];

    float Hpu_y[3]; // H_p * u_y
    Hpu_y[0] = Hp[0]*uy[0] + Hp[1]*uy[1] + Hp[2]*uy[2];
    Hpu_y[1] = Hp[1]*uy[0] + Hp[3]*uy[1] + Hp[4]*uy[2];
    Hpu_y[2] = Hp[2]*uy[0] + Hp[4]*uy[1] + Hp[5]*uy[2];

    // H_v elements:
    // Hv_xx = u_x^T H_p u_x
    out_H_v_packed[idx*3 + 0] = ux[0]*Hpu_x[0] + ux[1]*Hpu_x[1] + ux[2]*Hpu_x[2];
    // Hv_xy = u_x^T H_p u_y
    out_H_v_packed[idx*3 + 1] = ux[0]*Hpu_y[0] + ux[1]*Hpu_y[1] + ux[2]*Hpu_y[2];
    // Hv_yy = u_y^T H_p u_y
    out_H_v_packed[idx*3 + 2] = uy[0]*Hpu_y[0] + uy[1]*Hpu_y[1] + uy[2]*Hpu_y[2];
}

// Kernel for batch 2x2 solve
__global__ void batch_solve_2x2_system_kernel(
    int num_systems,
    const float* H_v_packed, // [N, 3] (H00, H01, H11)
    const float* g_v,        // [N, 2] (g0, g1)
    float damping,
    float step_scale,
    float* out_delta_v) {    // [N, 2]

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_systems) return;

    float H00 = H_v_packed[idx*3 + 0];
    float H01 = H_v_packed[idx*3 + 1];
    float H11 = H_v_packed[idx*3 + 2];

    float g0 = g_v[idx*2 + 0];
    float g1 = g_v[idx*2 + 1];

    // Add damping to diagonal
    H00 += damping;
    H11 += damping;

    float det = H00 * H11 - H01 * H01;

    // If det is too small, effectively no update or use gradient descent step
    if (abs(det) < 1e-8f) {
        out_delta_v[idx*2 + 0] = -step_scale * g0 / (H00 + 1e-6f); // Simplified fallback
        out_delta_v[idx*2 + 1] = -step_scale * g1 / (H11 + 1e-6f);
        return;
    }

    float inv_det = 1.f / det;

    // delta_v = - H_inv * g
    // H_inv = inv_det * [H11, -H01; -H01, H00]
    out_delta_v[idx*2 + 0] = -step_scale * inv_det * (H11 * g0 - H01 * g1);
    out_delta_v[idx*2 + 1] = -step_scale * inv_det * (-H01 * g0 + H00 * g1);
}

// Kernel for re-projecting delta_v to delta_p
__global__ void project_update_to_3d_kernel(
    int num_updates,
    const float* delta_v,          // [N, 2] (dvx, dvy)
    const float* means_3d_visible, // [N, 3] (Not strictly needed if U_k doesn't depend on p_k itself, but paper's U_k does via r_k)
    const float* view_matrix,      // [16]
    const float* cam_pos_world,    // [3]
    float* out_delta_p) {          // [N, 3]

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_updates) return;

    // Using the same simplified U_k = [cam_right, cam_up] as in projection kernel
    float ux[3] = {view_matrix[0], view_matrix[4], view_matrix[8]}; // Camera Right
    float uy[3] = {view_matrix[1], view_matrix[5], view_matrix[9]}; // Camera Up

    float dvx = delta_v[idx*2 + 0];
    float dvy = delta_v[idx*2 + 1];

    // delta_p = U_k * delta_v = u_x * dvx + u_y * dvy
    out_delta_p[idx*3 + 0] = ux[0] * dvx + uy[0] * dvy;
    out_delta_p[idx*3 + 1] = ux[1] * dvx + uy[1] * dvy;
    out_delta_p[idx*3 + 2] = ux[2] * dvx + uy[2] * dvy;
}

// --- Spherical Harmonics Basis Evaluation Kernel ---
// Based on gsplat's sh_coeffs_to_color_fast, but only computes basis values.
__global__ void eval_sh_basis_kernel(
    const int num_points, const int degree,
    const float* dirs, float* sh_basis_output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_points) return;
    float x = dirs[idx * 3 + 0]; float y = dirs[idx * 3 + 1]; float z = dirs[idx * 3 + 2];
    int num_sh_coeffs = (degree + 1) * (degree + 1);
    float* current_sh_output = sh_basis_output + idx * num_sh_coeffs;
    current_sh_output[0] = 0.2820947917738781f;
    if (degree == 0) return;
    current_sh_output[1] = -0.48860251190292f * y;
    current_sh_output[2] = 0.48860251190292f * z;
    current_sh_output[3] = -0.48860251190292f * x;
    if (degree == 1) return;
    float z2 = z * z; float fTmp0B = -1.092548430592079f * z;
    float fC1 = x * x - y * y; float fS1 = 2.f * x * y;
    current_sh_output[4] = 0.5462742152960395f * fS1;
    current_sh_output[5] = fTmp0B * y;
    current_sh_output[6] = (0.9461746957575601f * z2 - 0.3153915652525201f);
    current_sh_output[7] = fTmp0B * x;
    current_sh_output[8] = 0.5462742152960395f * fC1;
    if (degree == 2) return;
    float fTmp0C = -2.285228997322329f * z2 + 0.4570457994644658f;
    float fTmp1B = 1.445305721320277f * z;
    float fC2 = x * fC1 - y * fS1; float fS2 = x * fS1 + y * fC1;
    current_sh_output[9]  = -0.5900435899266435f * fS2;
    current_sh_output[10] = fTmp1B * fS1;
    current_sh_output[11] = fTmp0C * y;
    current_sh_output[12] = z * (1.865881662950577f * z2 - 1.119528997770346f);
    current_sh_output[13] = fTmp0C * x;
    current_sh_output[14] = fTmp1B * fC1;
    current_sh_output[15] = -0.5900435899266435f * fC2;
    if (degree == 3) return;
    float fTmp0D = z * (-4.683325804901025f * z2 + 2.007139630671868f);
    float fTmp1C = 3.31161143515146f * z2 - 0.47308734787878f;
    float fTmp2B = -1.770130769779931f * z;
    float fC3 = x * fC2 - y * fS2; float fS3 = x * fS2 + y * fC2;
    float pSH6_val = (0.9461746957575601f * z2 - 0.3153915652525201f);
    float pSH12_val = z * (1.865881662950577f * z2 - 1.119528997770346f);
    current_sh_output[16] = 0.6258357354491763f * fS3;
    current_sh_output[17] = fTmp2B * fS2;
    current_sh_output[18] = fTmp1C * fS1;
    current_sh_output[19] = fTmp0D * y;
    current_sh_output[20] = (1.984313483298443f * z * pSH12_val - 1.006230589874905f * pSH6_val);
    current_sh_output[21] = fTmp0D * x;
    current_sh_output[22] = fTmp1C * fC1;
    current_sh_output[23] = fTmp2B * fC2;
    current_sh_output[24] = 0.6258357354491763f * fC3;
}

// --- Kernel for batch 3x3 solve ---
__global__ void batch_solve_3x3_symmetric_system_kernel(
    int num_systems,
    const float* H_packed, const float* g,
    float damping, float* out_x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_systems) return;
    const float* Hp = &H_packed[idx * 6];
    const float* gp = &g[idx * 3];
    float* xp = &out_x[idx * 3];
    float a00 = Hp[0] + damping; float a01 = Hp[1];         float a02 = Hp[2];
    float a10 = Hp[1];         float a11 = Hp[3] + damping; float a12 = Hp[4];
    float a20 = Hp[2];         float a21 = Hp[4];         float a22 = Hp[5] + damping;
    float detA = a00 * (a11 * a22 - a12 * a21) -
                 a01 * (a10 * a22 - a12 * a20) +
                 a02 * (a10 * a21 - a11 * a20);
    if (abs(detA) < 1e-9f) {
        xp[0] = -gp[0] / (a00 + 1e-6f);
        xp[1] = -gp[1] / (a11 + 1e-6f);
        xp[2] = -gp[2] / (a22 + 1e-6f);
        return;
    }
    float invDetA = 1.0f / detA;
    xp[0] = invDetA * ((a11*a22 - a12*a21)*(-gp[0]) + (a02*a21 - a01*a22)*(-gp[1]) + (a01*a12 - a02*a11)*(-gp[2]));
    xp[1] = invDetA * ((a12*a20 - a10*a22)*(-gp[0]) + (a00*a22 - a02*a20)*(-gp[1]) + (a02*a10 - a00*a12)*(-gp[2]));
    xp[2] = invDetA * ((a10*a21 - a11*a20)*(-gp[0]) + (a01*a20 - a00*a21)*(-gp[1]) + (a00*a11 - a01*a10)*(-gp[2]));
}

// --- Kernel for batch 1x1 solve ---
__global__ void batch_solve_1x1_system_kernel(
    int num_systems,
    const float* H_scalar, const float* g_scalar,
    float damping, float* out_x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_systems) return;
    float h_val = H_scalar[idx];
    float g_val = g_scalar[idx];
    float h_damped = h_val + damping;
    if (abs(h_damped) < 1e-9f) {
        out_x[idx] = 0.0f;
    } else {
        out_x[idx] = -g_val / h_damped;
    }
}


// --- LAUNCHER FUNCTIONS ---

void NewtonKernels::compute_loss_derivatives_kernel_launcher(
    const torch::Tensor& rendered_image_tensor,
    const torch::Tensor& gt_image_tensor,
    float lambda_dssim,
    bool use_l2_loss_term,
    torch::Tensor& out_dL_dc_tensor,
    torch::Tensor& out_d2L_dc2_diag_tensor) {

    int H = rendered_image_tensor.size(0);
    int W = rendered_image_tensor.size(1);
    int C = rendered_image_tensor.size(2);
    int total_elements = H * W * C;

    const float* rendered_image_ptr = gs::torch_utils::get_const_data_ptr<float>(rendered_image_tensor);
    const float* gt_image_ptr = gs::torch_utils::get_const_data_ptr<float>(gt_image_tensor);
    float* out_dL_dc_ptr = gs::torch_utils::get_data_ptr<float>(out_dL_dc_tensor);
    float* out_d2L_dc2_diag_ptr = gs::torch_utils::get_data_ptr<float>(out_d2L_dc2_diag_tensor);

    // Create temporary tensors for L1/L2 parts
    auto tensor_opts = rendered_image_tensor.options();
    torch::Tensor dL_dc_l1l2 = torch::empty_like(rendered_image_tensor, tensor_opts);
    torch::Tensor d2L_dc2_diag_l1l2 = torch::empty_like(rendered_image_tensor, tensor_opts);

    // Calculate normalization factor
    const float N_pixels = static_cast<float>(H * W);
    const float inv_N_pixels = (N_pixels > 0) ? (1.0f / N_pixels) : 1.0f; // Avoid div by zero if H*W=0

    // Call kernel for L1/L2 derivatives
    compute_l1l2_loss_derivatives_kernel<<<GET_BLOCKS(total_elements), CUDA_NUM_THREADS>>>(
        rendered_image_ptr, gt_image_ptr, use_l2_loss_term, inv_N_pixels,
        gs::torch_utils::get_data_ptr<float>(dL_dc_l1l2),
        gs::torch_utils::get_data_ptr<float>(d2L_dc2_diag_l1l2),
        H, W, C
    );
    CUDA_CHECK(hipGetLastError());

    // --- SSIM Part ---
    // Constants for SSIM
    const float C1 = 0.01f * 0.01f;
    const float C2 = 0.03f * 0.03f;

    // Reshape and permute images for SSIM functions: [H,W,C] -> [1,C,H,W]
    torch::Tensor img1_bchw = rendered_image_tensor.unsqueeze(0).permute({0, 3, 1, 2}).contiguous();
    torch::Tensor img2_bchw = gt_image_tensor.unsqueeze(0).permute({0, 3, 1, 2}).contiguous();

    // Call fusedssim to get ssim_map and intermediate derivatives for backward pass
    // Need to include "kernels/ssim.cuh" for these C++ functions
    auto ssim_outputs = fusedssim(C1, C2, img1_bchw, img2_bchw, true /* train=true */);
    torch::Tensor ssim_map_bchw = std::get<0>(ssim_outputs);
    torch::Tensor dm_dmu1 = std::get<1>(ssim_outputs);
    torch::Tensor dm_dsigma1_sq = std::get<2>(ssim_outputs);
    torch::Tensor dm_dsigma12 = std::get<3>(ssim_outputs);

    // Define dL_s/d(ssim_map). Assuming L_s = DSSIM = (1 - SSIM)/2, so dL_s/d(ssim_map) = -0.5
    // The lambda_dssim is applied to the result of dL_s/dc.
    // If L_s is the loss term itself, then dL_s/d(map) is the derivative of that loss.
    // If the loss is L = (1-lambda)*L2 + lambda*DSSIM, then dL/d(DSSIM) = lambda.
    // And d(DSSIM)/d(SSIM_map) = -0.5. So dL/d(SSIM_map) = -0.5 * lambda.
    // However, the formulation L = L2 + lambda*L_S suggests lambda is a weight for L_S.
    // Let's assume L_S is DSSIM. Then the dL_s/dc we compute will be d(DSSIM)/dc.
    // The final dL/dc will be dL2/dc + lambda_dssim * d(DSSIM)/dc.
    // So, for d(DSSIM)/dc, we need d(DSSIM)/d(SSIM_map) = -0.5.
    torch::Tensor dL_dmap_tensor = torch::full_like(ssim_map_bchw, -0.5f);

    // Call fusedssim_backward to get d(SSIM_loss)/dc (which is d(DSSIM)/dc if dL_dmap is for DSSIM)
    torch::Tensor dL_dc_ssim_bchw = fusedssim_backward(
        C1, C2, img1_bchw, img2_bchw, dL_dmap_tensor, dm_dmu1, dm_dsigma1_sq, dm_dsigma12
    );

    // Permute dL_dc_ssim back to [H,W,C]
    torch::Tensor dL_dc_ssim_hwc_unnormalized = dL_dc_ssim_bchw.permute({0, 2, 3, 1}).squeeze(0).contiguous();
    torch::Tensor dL_dc_ssim_hwc_normalized = dL_dc_ssim_hwc_unnormalized * inv_N_pixels;

    // Combine derivatives: out_dL_dc = dL_dc_l1l2 + lambda_dssim * dL_dc_ssim_hwc_normalized
    // dL_dc_l1l2 is already normalized by inv_N_pixels inside its kernel
    out_dL_dc_tensor.copy_(dL_dc_l1l2 + lambda_dssim * dL_dc_ssim_hwc_normalized);

    // Set out_d2L_dc2_diag_tensor:
    // d2L_dc2_diag_l1l2 is already normalized by inv_N_pixels inside its kernel
    // As discussed, SSIM's second derivative d2L_s/dc2 is not computed by ssim.cu.
    // We assume it's effectively zero or handled by use_l2_for_hessian_L_term logic,
    // meaning only the L1/L2 part contributes to the d2L/dc2 term in Hessian assembly.
    out_d2L_dc2_diag_tensor.copy_(d2L_dc2_diag_l1l2);

    CUDA_CHECK(hipGetLastError()); // Check for errors from SSIM calls too
}


void NewtonKernels::compute_position_hessian_components_kernel_launcher(
    int H_img, int W_img, int C_img,
    int P_total,
    const float* means_3d_all, const float* scales_all, const float* rotations_all,
    const float* opacities_all, const float* shs_all,
    int sh_degree,
    int sh_coeffs_per_color_channel, // Changed from sh_coeffs_dim
    const float* view_matrix_ptr, // Already a pointer from C++
    const float* perspective_proj_matrix_ptr, // Changed from projection_matrix_for_jacobian
    const float* cam_pos_world_ptr, // Already a pointer from C++
    // Removed means_2d_render, depths_render, radii_render, P_render as kernel doesn't use them directly
    const torch::Tensor& visibility_mask_for_model_tensor,
    const float* dL_dc_pixelwise_ptr, // Already a pointer
    const float* d2L_dc2_diag_pixelwise_ptr, // Already a pointer
    int num_output_gaussians,
    float* H_p_output_packed_ptr, // Already a pointer
    float* grad_p_output_ptr,   // Already a pointer
    bool debug_prints_enabled
) {
    // Construct the output_index_map (mapping P_total index to dense output index)
    TORCH_CHECK(visibility_mask_for_model_tensor.defined(), "visibility_mask_for_model_tensor is not defined in launcher.");
    TORCH_CHECK(visibility_mask_for_model_tensor.scalar_type() == torch::kBool, "visibility_mask_for_model_tensor must be Bool type.");
    TORCH_CHECK(static_cast<int>(visibility_mask_for_model_tensor.size(0)) == P_total, "visibility_mask_for_model_tensor size mismatch with P_total.");

    torch::Tensor visibility_mask_cpu = visibility_mask_for_model_tensor.to(torch::kCPU).contiguous();
    const bool* cpu_visibility_ptr = visibility_mask_cpu.data_ptr<bool>();

    std::vector<int> output_index_map_cpu(P_total);
    int current_out_idx = 0;
    for(int i=0; i<P_total; ++i) {
        if(cpu_visibility_ptr[i]) {
            output_index_map_cpu[i] = current_out_idx++;
        } else {
            output_index_map_cpu[i] = -1;
        }
    }
    // AT_ASSERTM(current_out_idx == num_output_gaussians, "Mismatch in visible count for output_index_map"); // Good check but might fail if num_output_gaussians is pre-calculated slightly differently.

    torch::Tensor output_index_map_tensor = torch::tensor(output_index_map_cpu,
        torch::TensorOptions().dtype(torch::kInt).device(visibility_mask_for_model_tensor.device())); // Keep on same device
    const int* output_index_map_gpu = gs::torch_utils::get_const_data_ptr<int>(output_index_map_tensor, "output_index_map_tensor_in_launcher");
    const bool* visibility_mask_gpu_ptr = gs::torch_utils::get_const_data_ptr<bool>(visibility_mask_for_model_tensor, "visibility_mask_for_model_tensor_for_kernel");

    if (debug_prints_enabled && P_total > 0) { // Added P_total > 0 to avoid printing for empty scenes
         printf("[CUDA LAUNCHER] compute_position_hessian_components_kernel. P_total: %d, num_output_gaussians: %d, H/W/C: %d/%d/%d\n",
                P_total, num_output_gaussians, H_img, W_img, C_img);
    }

    compute_position_hessian_components_kernel<<<GET_BLOCKS(P_total), CUDA_NUM_THREADS>>>(
        H_img, W_img, C_img,
        P_total,
        means_3d_all,
        scales_all,
        rotations_all,
        opacities_all,
        shs_all,
        sh_degree,
        sh_coeffs_per_color_channel, // Use the new name
        view_matrix_ptr,
        perspective_proj_matrix_ptr, // Use the new name (4x4 P matrix)
        cam_pos_world_ptr,
        visibility_mask_gpu_ptr,
        dL_dc_pixelwise_ptr,
        d2L_dc2_diag_pixelwise_ptr,
        num_output_gaussians,
        H_p_output_packed_ptr,
        grad_p_output_ptr,
        output_index_map_gpu,
        debug_prints_enabled
    );
    CUDA_CHECK(hipGetLastError());
}

void NewtonKernels::project_position_hessian_gradient_kernel_launcher(
    int num_visible_gaussians,
    const float* H_p_packed_input, const float* grad_p_input,
    const float* means_3d_visible, const float* view_matrix,
    const float* cam_pos_world,
    float* out_H_v_packed, float* out_grad_v ) {

    project_position_hessian_gradient_kernel<<<GET_BLOCKS(num_visible_gaussians), CUDA_NUM_THREADS>>>(
        num_visible_gaussians, H_p_packed_input, grad_p_input, means_3d_visible,
        view_matrix, cam_pos_world, out_H_v_packed, out_grad_v
    );
    CUDA_CHECK(hipGetLastError());
}

void NewtonKernels::batch_solve_2x2_system_kernel_launcher(
    int num_systems,
    const float* H_v_packed, const float* g_v, float damping, float step_scale,
    float* out_delta_v ) {

    batch_solve_2x2_system_kernel<<<GET_BLOCKS(num_systems), CUDA_NUM_THREADS>>>(
        num_systems, H_v_packed, g_v, damping, step_scale, out_delta_v
    );
    CUDA_CHECK(hipGetLastError());
}

void NewtonKernels::project_update_to_3d_kernel_launcher(
    int num_updates,
    const float* delta_v, const float* means_3d_visible,
    const float* view_matrix, const float* cam_pos_world,
    float* out_delta_p ) {

    project_update_to_3d_kernel<<<GET_BLOCKS(num_updates), CUDA_NUM_THREADS>>>(
        num_updates, delta_v, means_3d_visible, view_matrix, cam_pos_world, out_delta_p
    );
    CUDA_CHECK(hipGetLastError());
}

// --- Definitions for Scale Optimization Launchers (Stubs) ---

void NewtonKernels::compute_scale_hessian_gradient_components_kernel_launcher(
    int H_img, int W_img, int C_img,
    int P_total,
    const torch::Tensor& means_all,
    const torch::Tensor& scales_all,
    const torch::Tensor& rotations_all,
    const torch::Tensor& opacities_all,
    const torch::Tensor& shs_all,
    int sh_degree,
    const torch::Tensor& view_matrix,
    const torch::Tensor& K_matrix,
    const torch::Tensor& cam_pos_world,
    const gs::RenderOutput& render_output,
    const torch::Tensor& visible_indices,
    const torch::Tensor& dL_dc_pixelwise,
    const torch::Tensor& d2L_dc2_diag_pixelwise,
    torch::Tensor& out_H_s_packed,
    torch::Tensor& out_g_s
    // bool debug_prints_enabled // TODO: Add this if needed
) {
    // TODO: Pass debug_prints_enabled if options_.debug_print_shapes is to be respected here
    // if (debug_prints_enabled) {
    //     printf("[STUB KERNEL LAUNCHER] compute_scale_hessian_gradient_components_kernel_launcher called.\n");
    // }
    // This function would:
    // 1. Prepare raw pointers from all input tensors.
    // 2. Launch one or more CUDA kernels to compute ∂c/∂s_k, ∂²c/∂s_k², and then accumulate
    //    H_s_k and g_s_k for each visible Gaussian.
    // For now, it does nothing, out_H_s_packed and out_g_s remain as initialized (e.g. zeros).
}

void NewtonKernels::batch_solve_3x3_system_kernel_launcher(
    int num_systems,
    const torch::Tensor& H_s_packed,
    const torch::Tensor& g_s,
    float damping,
    torch::Tensor& out_delta_s
    // bool debug_prints_enabled // TODO: Add this if needed
) {
    // TODO: Pass debug_prints_enabled if options_.debug_print_shapes is to be respected here
    // if (debug_prints_enabled) {
    //    printf("[STUB KERNEL LAUNCHER] batch_solve_3x3_system_kernel_launcher called for %d systems.\n", num_systems);
    // }
    // This function would:
    // 1. Prepare raw pointers.
    // 2. Launch a CUDA kernel to solve N independent 3x3 systems: H_s * Δs = -g_s.
    //    (H_s is symmetric, so 6 unique elements from H_s_packed).
    TORCH_CHECK(H_s_packed.defined() && H_s_packed.dim() == 2 && H_s_packed.size(1) == 6, "H_s_packed shape must be [N, 6]");
    TORCH_CHECK(g_s.defined() && g_s.dim() == 2 && g_s.size(1) == 3, "g_s shape must be [N, 3]");
    TORCH_CHECK(out_delta_s.defined() && out_delta_s.dim() == 2 && out_delta_s.size(1) == 3, "out_delta_s shape must be [N, 3]");
    TORCH_CHECK(H_s_packed.size(0) == num_systems && g_s.size(0) == num_systems && out_delta_s.size(0) == num_systems, "Batch size mismatch");
    TORCH_CHECK(H_s_packed.is_cuda() && g_s.is_cuda() && out_delta_s.is_cuda(), "All tensors must be CUDA tensors");
    TORCH_CHECK(H_s_packed.is_contiguous() && g_s.is_contiguous() && out_delta_s.is_contiguous(), "All tensors must be contiguous");

    if (num_systems == 0) return;

    const float* H_ptr = gs::torch_utils::get_const_data_ptr<float>(H_s_packed, "H_s_packed");
    const float* g_ptr = gs::torch_utils::get_const_data_ptr<float>(g_s, "g_s");
    float* delta_s_ptr = gs::torch_utils::get_data_ptr<float>(out_delta_s, "out_delta_s");

    batch_solve_3x3_symmetric_system_kernel<<<GET_BLOCKS(num_systems), CUDA_NUM_THREADS>>>(
        num_systems,
        H_ptr,
        g_ptr,
        damping,
        delta_s_ptr
    );
    CUDA_CHECK(hipGetLastError());
}

// --- Definitions for Rotation Optimization Launchers (Stubs) ---

void NewtonKernels::compute_rotation_hessian_gradient_components_kernel_launcher(
    int H_img, int W_img, int C_img,
    int P_total,
    const torch::Tensor& means_all,
    const torch::Tensor& scales_all,
    const torch::Tensor& rotations_all,
    const torch::Tensor& opacities_all,
    const torch::Tensor& shs_all,
    int sh_degree,
    const torch::Tensor& view_matrix,
    const torch::Tensor& K_matrix,
    const torch::Tensor& cam_pos_world,
    const torch::Tensor& r_k_vecs,
    const gs::RenderOutput& render_output,
    const torch::Tensor& visible_indices,
    const torch::Tensor& dL_dc_pixelwise,
    const torch::Tensor& d2L_dc2_diag_pixelwise,
    torch::Tensor& out_H_theta,
    torch::Tensor& out_g_theta) {
    // This function would:
    // 1. Prepare raw pointers from input tensors.
    // 2. Launch CUDA kernel(s) to compute ∂c/∂θ_k, ∂²c/∂θ_k², and then accumulate
    //    H_θ_k and g_θ_k for each visible Gaussian, using r_k as rotation axis.
    // For now, it does nothing; out_H_theta and out_g_theta remain as initialized.
    // if (options_debug_print_shapes_can_be_passed_here) {
    //     printf("[STUB KERNEL LAUNCHER] compute_rotation_hessian_gradient_components_kernel_launcher called.\n");
    // }
}

void NewtonKernels::batch_solve_1x1_system_kernel_launcher(
    int num_systems,
    const torch::Tensor& H_theta,
    const torch::Tensor& g_theta,
    float damping,
    torch::Tensor& out_delta_theta) {
    // This function would:
    // 1. Prepare raw pointers.
    // 2. Launch a CUDA kernel to solve N independent 1x1 systems:
    //    (H_theta_k + damping) * Δθ_k = -g_theta_k  => Δθ_k = -g_theta_k / (H_theta_k + damping)
    // For now, it does nothing; out_delta_theta remains as initialized.
    // The calling C++ code in NewtonOptimizer currently has a placeholder for this.
    // if (options_debug_print_shapes_can_be_passed_here) {
    //     printf("[STUB KERNEL LAUNCHER] batch_solve_1x1_system_kernel_launcher called for %d systems.\n", num_systems);
    // }
    TORCH_CHECK(H_theta.defined() && H_theta.size(0) == num_systems, "H_theta size mismatch");
    TORCH_CHECK(g_theta.defined() && g_theta.size(0) == num_systems, "g_theta size mismatch");
    TORCH_CHECK(out_delta_theta.defined() && out_delta_theta.size(0) == num_systems, "out_delta_theta size mismatch");
    TORCH_CHECK(H_theta.is_cuda() && g_theta.is_cuda() && out_delta_theta.is_cuda(), "All tensors must be CUDA tensors");
    // Assuming tensors can be [N] or [N,1].contiguous() makes them effectively [N] for data_ptr.
    // If they must be [N,1], ensure contiguity after potential reshape.
    // For simplicity, assume they are already prepared as contiguous (e.g. after .contiguous() call if reshaped from [N,1])

    if (num_systems == 0) return;

    const float* H_ptr = gs::torch_utils::get_const_data_ptr<float>(H_theta.contiguous(), "H_theta");
    const float* g_ptr = gs::torch_utils::get_const_data_ptr<float>(g_theta.contiguous(), "g_theta");
    float* delta_theta_ptr = gs::torch_utils::get_data_ptr<float>(out_delta_theta.contiguous(), "out_delta_theta"); // Ensure contiguous for output too

    batch_solve_1x1_system_kernel<<<GET_BLOCKS(num_systems), CUDA_NUM_THREADS>>>(
        num_systems,
        H_ptr,
        g_ptr,
        damping,
        delta_theta_ptr
    );
    CUDA_CHECK(hipGetLastError());
}

// --- Definitions for Opacity Optimization Launchers (Stubs) ---

void NewtonKernels::compute_opacity_hessian_gradient_components_kernel_launcher(
    int H_img, int W_img, int C_img,
    int P_total,
    const torch::Tensor& means_all,
    const torch::Tensor& scales_all,
    const torch::Tensor& rotations_all,
    const torch::Tensor& opacities_all,
    const torch::Tensor& shs_all,
    int sh_degree,
    const torch::Tensor& view_matrix,
    const torch::Tensor& K_matrix,
    const torch::Tensor& cam_pos_world,
    const gs::RenderOutput& render_output,
    const torch::Tensor& visible_indices,
    const torch::Tensor& dL_dc_pixelwise,
    const torch::Tensor& d2L_dc2_diag_pixelwise,
    torch::Tensor& out_H_sigma_base,
    torch::Tensor& out_g_sigma_base) {
    // This function would:
    // 1. Prepare raw pointers from input tensors.
    // 2. Launch CUDA kernel(s) to compute ∂c/∂σ_k. The paper states ∂²c/∂σ_k² = 0.
    //    The formula for ∂c/∂σ_k involves terms like G_k, accumulated alpha from prior Gaussians,
    //    the Gaussian's own color c_k, and the color accumulated from Gaussians behind it.
    //    This requires careful handling of sorted Gaussians and their blended contributions.
    // 3. Accumulate H_σ_base_k and g_σ_base_k for each visible Gaussian:
    //    g_σ_base_k = sum_pixels [ (∂c/∂σ_k)ᵀ ⋅ (dL/dc) ]
    //    H_σ_base_k = sum_pixels [ (∂c/∂σ_k)ᵀ ⋅ (d²L/dc²) ⋅ (∂c/∂σ_k) ]
    // For now, it does nothing; out_H_sigma_base and out_g_sigma_base remain as initialized (e.g., zeros).
    // if (options_debug_print_shapes_can_be_passed_here) { // Assuming a debug flag could be passed
    //     printf("[STUB KERNEL LAUNCHER] compute_opacity_hessian_gradient_components_kernel_launcher called.\n");
    // }
    // This is a stub. A real implementation needs a kernel.
    // For now, to avoid linker errors if called, we ensure outputs are zeroed if they are not already.
    if (out_H_sigma_base.defined()) out_H_sigma_base.zero_();
    if (out_g_sigma_base.defined()) out_g_sigma_base.zero_();
}

// --- Definitions for SH (Color) Optimization Launchers (Stubs) ---

torch::Tensor NewtonKernels::compute_sh_bases_kernel_launcher(
    int sh_degree,
    const torch::Tensor& normalized_view_vectors) {
    // This function would:
    // 1. Prepare raw pointers.
    // 2. Launch a CUDA kernel to evaluate SH basis functions B_k(r_k) for each view vector.
    //    Output shape: [N_vis, (sh_degree+1)^2]
    // For now, returns empty tensor or zeros of correct shape.
    // if (options_debug_print_shapes_can_be_passed_here) {
    //     printf("[STUB KERNEL LAUNCHER] compute_sh_bases_kernel_launcher called.\n");
    // }
    TORCH_CHECK(normalized_view_vectors.defined(), "normalized_view_vectors must be defined.");
    TORCH_CHECK(normalized_view_vectors.dim() == 2 && normalized_view_vectors.size(1) == 3,
                "normalized_view_vectors must have shape [N, 3]. Got ", normalized_view_vectors.sizes());
    TORCH_CHECK(normalized_view_vectors.is_cuda(), "normalized_view_vectors must be a CUDA tensor.");
    TORCH_CHECK(normalized_view_vectors.is_contiguous(), "normalized_view_vectors must be contiguous.");
    TORCH_CHECK(sh_degree >= 0 && sh_degree <= 4, "sh_degree must be between 0 and 4. Got ", sh_degree);

    const int num_points = normalized_view_vectors.size(0);
    if (num_points == 0) {
        return torch::empty({0, (sh_degree + 1) * (sh_degree + 1)}, normalized_view_vectors.options());
    }

    const int num_sh_coeffs = (sh_degree + 1) * (sh_degree + 1);
    torch::Tensor sh_bases_tensor = torch::empty({num_points, num_sh_coeffs}, normalized_view_vectors.options());

    const float* dirs_ptr = gs::torch_utils::get_const_data_ptr<float>(normalized_view_vectors, "normalized_view_vectors");
    float* sh_basis_output_ptr = gs::torch_utils::get_data_ptr<float>(sh_bases_tensor, "sh_bases_tensor");

    eval_sh_basis_kernel<<<GET_BLOCKS(num_points), CUDA_NUM_THREADS>>>(
        num_points,
        sh_degree,
        dirs_ptr,
        sh_basis_output_ptr
    );
    CUDA_CHECK(hipGetLastError());

    return sh_bases_tensor;
}

void NewtonKernels::compute_sh_hessian_gradient_components_kernel_launcher(
    int H_img, int W_img, int C_img,
    int P_total,
    const torch::Tensor& means_all,
    const torch::Tensor& scales_all,
    const torch::Tensor& rotations_all,
    const torch::Tensor& opacities_all,
    const torch::Tensor& shs_all,
    int sh_degree,
    const torch::Tensor& sh_bases_values,
    const torch::Tensor& view_matrix,
    const torch::Tensor& K_matrix,
    const gs::RenderOutput& render_output,
    const torch::Tensor& visible_indices,
    const torch::Tensor& dL_dc_pixelwise,
    const torch::Tensor& d2L_dc2_diag_pixelwise,
    torch::Tensor& out_H_ck_diag,
    torch::Tensor& out_g_ck) {
    // This function would:
    // 1. Prepare raw pointers.
    // 2. Launch CUDA kernel(s) to compute Jacobian J_sh = ∂c_pixel/∂c_k (using sh_bases_values)
    //    and then accumulate H_ck_base and g_ck_base.
    //    Paper: ∂c_R/∂c_{k,R} = sum_{gaussians} G_k σ_k (Π(1-G_jσ_j)) B_{k,R}
    //    If ∂²c_R/∂c_{k,R}² (direct part) = 0, then Hessian is J_sh^T * (d2L/dc2) * J_sh
    // For now, it does nothing. out_H_ck_diag and out_g_ck remain as initialized.
    // if (options_debug_print_shapes_can_be_passed_here) {
    //     printf("[STUB KERNEL LAUNCHER] compute_sh_hessian_gradient_components_kernel_launcher called.\n");
    // }
    // This is a stub. A real implementation needs a kernel.
    // For now, to avoid linker errors if called, we ensure outputs are zeroed.
    if (out_H_ck_diag.defined()) out_H_ck_diag.zero_();
    if (out_g_ck.defined()) out_g_ck.zero_();
}

[end of kernels/newton_kernels.cu]
